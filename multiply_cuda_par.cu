
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

int N, *A, *B, *C;

void createMatrix(int **matrix) {
    *matrix = (int*)malloc(N*N * sizeof(int));
}

void fillMatrix(int *matrix) {
    for(int i = 0; i < N * N; i++) {
        matrix[i] = rand()%10 + 1;
    }
}

void init() {
    createMatrix(&A);
    fillMatrix(A);
    createMatrix(&B);
    fillMatrix(B);
    createMatrix(&C);
}

__global__ void multiply(int *A, int *B, int *C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int tmp = 0;

    if(row < N && col < N) {
        int shift = row * N;
        for(int k = 0; k < N; k++)
            tmp += A[shift + k] * B[k * N + col];
        C[shift + col] = tmp;
    }
}

void printMatrix(int *matrix) {
    for(int i = 0; i < N * N; i++) {
        if(i % N == 0 && i != 0) {
            printf("\n");
        }
        printf("%d ", matrix[i]);
    }
    printf("\n");
}

int main(int argc, char* argv[]){
    switch(argc) {
        case 2:
            sscanf(argv[1], "%d", &N);
            break;
        default:
            printf("Wrong parameters. Closing program...\n");
            return 0;
    }
    if(N < 2) {
        printf("Wrong parameters. Closing program...\n");
        return 0;
    }

    init();

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    multiply<<<1, N*N>>>(A, B, C, N);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float msecTotal = 0.0f;
    hipEventElapsedTime(&msecTotal, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("MATRIX: %dx%d\nTIME: %f\n", N, N, msecTotal);
    // printMatrix(C);
}